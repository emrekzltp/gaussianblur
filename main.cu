#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>
#include <ctime>
#include <hip/hip_runtime.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

struct Pixel {
    unsigned char r, g, b;
};

__global__ void applyGaussianBlurKernel(Pixel* d_input, Pixel* d_output, float* d_kernel, int width, int height, int kernelSize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int halfSize = kernelSize / 2;

    if (x < width && y < height) {
        float rSum = 0.0f, gSum = 0.0f, bSum = 0.0f;
        for (int ky = -halfSize; ky <= halfSize; ++ky) {
            for (int kx = -halfSize; kx <= halfSize; ++kx) {
                int nx = min(max(x + kx, 0), width - 1);
                int ny = min(max(y + ky, 0), height - 1);
                float kernelValue = d_kernel[(ky + halfSize) * kernelSize + (kx + halfSize)];
                Pixel p = d_input[ny * width + nx];
                rSum += kernelValue * p.r;
                gSum += kernelValue * p.g;
                bSum += kernelValue * p.b;
            }
        }
        d_output[y * width + x].r = static_cast<unsigned char>(rSum);
        d_output[y * width + x].g = static_cast<unsigned char>(gSum);
        d_output[y * width + x].b = static_cast<unsigned char>(bSum);
    }
}

std::vector<std::vector<float>> generateGaussianKernel(int kernelSize, float sigma) {
    std::vector<std::vector<float>> kernel(kernelSize, std::vector<float>(kernelSize));
    float sum = 0.0f;
    int halfSize = kernelSize / 2;
    float twoSigmaSquare = 2.0f * sigma * sigma;

    for (int x = -halfSize; x <= halfSize; ++x) {
        for (int y = -halfSize; y <= halfSize; ++y) {
            float exponent = -(x * x + y * y) / twoSigmaSquare;
            kernel[x + halfSize][y + halfSize] = exp(exponent) / (M_PI * twoSigmaSquare);
            sum += kernel[x + halfSize][y + halfSize];
        }
    }

    for (int i = 0; i < kernelSize; ++i) {
        for (int j = 0; j < kernelSize; ++j) {
            kernel[i][j] /= sum;
        }
    }
    return kernel;
}

int main() {
    try {
        std::string inputPath  = "./ex_pics/doggo.jpg";
        std::string outputPath = "./ex_pics/doggo_blurred.png";
        int width, height, channels;

        unsigned char* imageData = stbi_load(inputPath.c_str(), &width, &height, &channels, 3);
        if (!imageData) {
            throw std::runtime_error("Failed to load image");
        }

        std::vector<Pixel> image(width * height);
        for (int i = 0; i < width * height; i++) {
            image[i].r = imageData[i * 3];
            image[i].g = imageData[i * 3 + 1];
            image[i].b = imageData[i * 3 + 2];
        }
        stbi_image_free(imageData);

        int kernelSize;
        float sigma;

        std::cout << "Enter the kernel size >1 (suggested 15): ";
        std::cin >> kernelSize;
        if (kernelSize <= 1 || kernelSize % 2 == 0) {
            std::cerr << "Kernel size must be an odd number greater than 1." << std::endl;
            return -1;
        }

        std::cout << "Enter the sigma value (suggested 5): ";
        std::cin >> sigma;

        auto kernel = generateGaussianKernel(kernelSize, sigma);
        std::vector<float> flatKernel(kernelSize * kernelSize);
        for (int i = 0; i < kernelSize; i++) {
            for (int j = 0; j < kernelSize; j++) {
                flatKernel[i * kernelSize + j] = kernel[i][j];
            }
        }

        Pixel *d_input, *d_output;
        float *d_kernel;
        hipMalloc(&d_input, width * height * sizeof(Pixel));
        hipMalloc(&d_output, width * height * sizeof(Pixel));
        hipMalloc(&d_kernel, kernelSize * kernelSize * sizeof(float));

        hipMemcpy(d_input, image.data(), width * height * sizeof(Pixel), hipMemcpyHostToDevice);
        hipMemcpy(d_kernel, flatKernel.data(), kernelSize * kernelSize * sizeof(float), hipMemcpyHostToDevice);

        dim3 blockSize(16, 16);
        dim3 gridSize((width + 15) / 16, (height + 15) / 16);

        std::clock_t start = std::clock();
        applyGaussianBlurKernel<<<gridSize, blockSize>>>(d_input, d_output, d_kernel, width, height, kernelSize);
        hipDeviceSynchronize();
        std::clock_t end = std::clock();

        std::vector<Pixel> blurredImage(width * height);
        hipMemcpy(blurredImage.data(), d_output, width * height * sizeof(Pixel), hipMemcpyDeviceToHost);

        hipFree(d_input);
        hipFree(d_output);
        hipFree(d_kernel);

        std::vector<unsigned char> outputData(width * height * 3);
        for (int i = 0; i < width * height; i++) {
            outputData[i * 3] = blurredImage[i].r;
            outputData[i * 3 + 1] = blurredImage[i].g;
            outputData[i * 3 + 2] = blurredImage[i].b;
        }

        stbi_write_png(outputPath.c_str(), width, height, 3, outputData.data(), width * 3);
        double elapsedSeconds = static_cast<double>(end - start) / CLOCKS_PER_SEC;
        std::cout << "Blurred image saved to: " << outputPath << std::endl;
        std::cout << "Time taken for Gaussian blur: " << elapsedSeconds << " seconds." << std::endl;
    } catch (const std::exception& ex) {
        std::cerr << "Error: " << ex.what() << std::endl;
        return -1;
    }
    return 0;
}

